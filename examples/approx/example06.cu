
#include <hip/hip_runtime.h>
//saxpy,data,ctrl
//j, k
/*
Example 6: variables `j` and `k` come from an array and are used in a conditional.
Additionally, `k` is also used to index array `y`.

Approximate variables are `x[i]` and `x[i + 1]`.
*/
__global__ void saxpy(int n, float a, float *x, float *y) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = x[i];
  int k = x[i + 1];
  float sum = 0.0f;
  if (j < k) {
    y[k] = sum;
  }
}
