
#include <hip/hip_runtime.h>
__shared__ float y[1024];

__global__
void saxpy(int n, float a, float *x)
{
  if (n < 4) {
    y[10 + 2 * threadIdx.x] = a;
  } else {
    y[4 * threadIdx.x] = a;
  }
}
