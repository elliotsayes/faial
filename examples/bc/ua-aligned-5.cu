
#include <hip/hip_runtime.h>
// pass
// --blockDim=[32,32,1] --gridDim=2

__global__
void saxpy(int n, float a, float *x, float *y)
{
  if (threadIdx.x < 16) // half of the warp is enabled
  y[n + threadIdx.x] = a; // half of the cost + 1, cost: 3
}
