
#include <hip/hip_runtime.h>
__shared__ float y[1024];

__global__
void saxpy(int n, float a, float *x)
{
      for (int i = 0; i < n; i++) {
          y[2 * threadIdx.x] = a*x[i];
      }
      for (int j = 0; j < n; j++) {
          y[2 * threadIdx.x] = a*x[j];
      }
      for (int k = 0; k < n; k++) {
          y[2 * threadIdx.x] = a*x[k];
      }
}
