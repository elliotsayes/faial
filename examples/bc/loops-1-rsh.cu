
#include <hip/hip_runtime.h>
__shared__ float y[1024];

__global__
void saxpy(int n, float a, float *x)
{
  for (int i = 512; i >= 1; i >>= 1) {
    y[32 * threadIdx.x] = a*x[i];
  }
}
