
#include <hip/hip_runtime.h>
__shared__ float y[1024];

__global__
void saxpy(int n, float a, float *x)
{
  for (int i = 0; i < threadIdx.x + n; i++) {
      y[2 * threadIdx.x] = a*x[i];
  }
}
