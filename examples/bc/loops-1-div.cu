
#include <hip/hip_runtime.h>
__shared__ float y[1024];

__global__
void saxpy(int n, float a, float *x)
{
  for (int i = 512; i >= 1; i /= 2) {
    y[2 * threadIdx.x] = a*x[i];
  }
}
