
#include <hip/hip_runtime.h>
// pass
// --blockDim=[1,32,1]
__shared__ float y[1024];

__global__
void saxpy(int n, float a, float *x)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  // Racuda: 1
  // faial-bc: 1
  y[2 * threadIdx.y] = a*x[i];
}
