
#include <hip/hip_runtime.h>
// pass
// --blockDim=[1024,1,1] --gridDim=2
__shared__ float y[1024];

__device__ int f(int);

__global__
void saxpy(int n, float a, float *x)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int idx1 = x[i];
  int idx2 = f(threadIdx.x);
  y[idx1] = a;
  y[idx2] = a;
  y[n] = a;
}
