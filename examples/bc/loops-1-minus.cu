
#include <hip/hip_runtime.h>
__shared__ float y[1024];

__global__
void saxpy(int n, float a, float *x)
{
  for (int i = n; i >= 0; i--) {
    y[2 * threadIdx.x] = a*x[i];
  }
}
