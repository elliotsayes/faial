
#include <hip/hip_runtime.h>
__shared__ float y[1024];

__global__
void saxpy(int n, int m, float a, float *x)
{
  /*
  for (int i = 0; i < n; i += 5) {
    y[2 * threadIdx.x] = a*x[i];
  }*/
  for (int j = 0; j < n; j += m) {
    y[2 * threadIdx.x] = a*x[j];
  }
}
