
#include <hip/hip_runtime.h>
// pass
// --blockDim=[32,32,1] --gridDim=2

__global__
void saxpy(int n, float a, float *x, float *y)
{
  y[n + threadIdx.x] = a; // cost: 5
}
