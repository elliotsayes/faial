
#include <hip/hip_runtime.h>
__shared__ float y[1024];

__global__
void saxpy(int n, float a, float *x)
{
  for (int i = 0; i < n; i++) {
    for (int j = 1; j < n; j *= 2) {
      y[2 * threadIdx.x] = a*x[i];
    }
  }
}
