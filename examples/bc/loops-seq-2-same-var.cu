
#include <hip/hip_runtime.h>
__shared__ float y[1024];

__global__
void saxpy(int n, float a, float *x)
{
      for (int i = 0; i < n; i++) {
          y[2 * threadIdx.x] = a*x[i];
      }
      for (int i = 0; i < n; i++) {
          y[2 * threadIdx.x] = a*x[i];
      }
}
