
#include <hip/hip_runtime.h>
__shared__ float y[1024];

__global__
void saxpy(int n, float a, float *x)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  y[16 * threadIdx.x] = a*x[i];
}
