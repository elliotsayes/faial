
#include <hip/hip_runtime.h>
// pass
// --blockDim=[32,32,1] --gridDim=2

__global__
void saxpy(int n, float a, float *x, float *y)
{
  y[32*n*threadIdx.x] = a;
}
