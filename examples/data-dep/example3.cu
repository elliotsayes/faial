
#include <hip/hip_runtime.h>
// saxpy,ind,ctrl
// data flows from array to lower bound of loop
__global__ void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = x[i];
  for (int k = j; k < n; k++) {
    y[i] = y[i + 1];
  }
}
