
#include <hip/hip_runtime.h>
// control-dep, racy
__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (x[i]) y[i + 1] = a*x[i];
  y[i] = a*x[i];
}