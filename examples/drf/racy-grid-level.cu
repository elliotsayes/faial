
#include <hip/hip_runtime.h>
//pass
//--blockDim=2024 --gridDim=2024
// This example is only racy at the grid level.
__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = threadIdx.x;
  __syncthreads();
  if (i < n) y[i] = a*x[i] + y[i];
}
