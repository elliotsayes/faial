
#include <hip/hip_runtime.h>
__shared__ float y[1024][1024];

__global__
void saxpy(int n, float a, float *x)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i][i] = a*x[i] + y[i + 1][i + 1];
}
