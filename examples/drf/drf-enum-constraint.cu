
#include <hip/hip_runtime.h>
enum GridSamplerInterpolationMode { BILINEAR = 0, NEAREST, BICUBIC };

__global__
void saxpy(float a, float *x, float *y, GridSamplerInterpolationMode mode)
{
  int i = threadIdx.x;
  if (i < 4) {
    y[i + 4] = a*x[i] + y[i + mode];
  }
}
