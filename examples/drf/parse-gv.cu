
#include <hip/hip_runtime.h>
// pass
// --gridDim=[2,2,2]

// This example is only DRF when the tool parses the command line
// parameters.
__global__
void saxpy(int n, float a, float *x, float *y, int k)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) {
    y[i + 2] = a*x[i] +
      y[i + gridDim.x] +
      y[i + gridDim.y] +
      y[i + gridDim.z];
  }
}
