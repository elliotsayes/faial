
#include <hip/hip_runtime.h>
__global__

void saxpy(int n, float a, float *x, float *y)
{
  extern __shared__ float sram[];
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  float *z = sram;
  if (i < n) sram[i] = a*x[i] + z[i + 1];
}
