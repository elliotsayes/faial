
#include <hip/hip_runtime.h>
__global__
void saxpy_racefree(int n, float a, float *shmem, float *y)
{
unsigned tid = threadIdx.x;
int v = 0;
v += shmem[tid+16]; __syncwarp();
shmem[tid] = v; __syncwarp();
v += shmem[tid+8]; __syncwarp();
shmem[tid] = v; __syncwarp();
v += shmem[tid+4]; __syncwarp();
shmem[tid] = v; __syncwarp();
v += shmem[tid+2]; __syncwarp();
shmem[tid] = v; __syncwarp();
v += shmem[tid+1]; __syncwarp();
shmem[tid] = v;
}
