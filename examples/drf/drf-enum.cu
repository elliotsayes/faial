
#include <hip/hip_runtime.h>
enum GridSamplerInterpolationMode { BILINEAR = 0, NEAREST, BICUBIC };

__global__
void saxpy(float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  y[i] = a*x[i] + y[i + BILINEAR];
}
