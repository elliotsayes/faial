
#include <hip/hip_runtime.h>
__global__

void saxpy(int n, float a, float *x, float *y)
{
  extern __shared__ float sram[];
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  float *z = &sram[i * 3];
  if (i < n) sram[i * 3] = a*x[i] + z[3];
}
