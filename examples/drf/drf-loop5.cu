
#include <hip/hip_runtime.h>
__global__
void saxpy(unsigned int n, float a, float *x, float *y)
{
  float f = 0.0f;
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  y[0] = 0; // benign data-race
  for (int j = n; j > 0; j--) {
    f += y[j];
  }
  y[i + n + 1] = a*x[i] + f;
}
