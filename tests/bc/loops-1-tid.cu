
#include <hip/hip_runtime.h>
__shared__ float y[1024];

__global__
void saxpy(int n, float a, float *x)
{
  for (int i = threadIdx.x; i < n; i++) {
      y[2 * threadIdx.x] = a*x[i];
  }
}
