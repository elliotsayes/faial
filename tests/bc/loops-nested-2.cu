
#include <hip/hip_runtime.h>
__shared__ float y[1024];

__global__
void saxpy(int n, float a, float *x)
{
  for (int i = 0; i < n; i++) {
    for (int j = i; j < n; j++) {
      y[2 * threadIdx.x] = a*x[i];
    }
  }

  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      y[2 * threadIdx.x] = a*x[i];
    }
  }
}
